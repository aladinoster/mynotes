#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vecAddKernel(float *A, float *B, float *C, int n) {
  int i = threadIdx.x + blockDim.x * blockIdx.x;
  if (i < n) {
    C[i] = A[i] + B[i];
  }
}

void vecAddCPU(float *A, float *B, float *C, int n) {
  for (int i = 0; i < n; i++)
    C[i] = A[i] + B[i];
}

void vecAddCUDA(float *A_h, float *B_h, float *C_h, int n) {
  float *A_d, *B_d, *C_d;
  size_t size = n * sizeof(float);

  // Allocate device memory
  hipMalloc((void **)&A_d, size);
  hipMalloc((void **)&B_d, size);
  hipMalloc((void **)&C_d, size);

  // Copy host arrays to device
  hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

  // Launch kernel
  int blockSize = 256;
  int numBlocks = (n + blockSize - 1) / blockSize;
  vecAddKernel<<<numBlocks, blockSize>>>(A_d, B_d, C_d, n);

  // Copy result back to host
  hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(A_d);
  hipFree(B_d);
  hipFree(C_d);
}

int main() {
  int n = 1000; // Example size
  float *A = new float[n];
  float *B = new float[n];
  float *C = new float[n];

  // Initialize input arrays
  for (int i = 0; i < n; i++) {
    A[i] = i;
    B[i] = i * 2;
  }

  // Call CUDA vector addition
  vecAddCUDA(A, B, C, n);

  // Verify results
  for (int i = 0; i < n; i++) {
    if (C[i] != A[i] + B[i]) {
      std::cerr << "Error at index " << i << std::endl;
      break;
    }
  }

  std::cout << "Vector addition completed successfully!" << std::endl;

  delete[] A;
  delete[] B;
  delete[] C;

  return 0;
}
